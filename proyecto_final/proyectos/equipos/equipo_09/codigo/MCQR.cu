#include "hip/hip_runtime.h"
/* Equipo 9
 *  Factorización QR utilizando CUDA, cuSolver
 *  Para compilar se debe ejecutar:
 *  make 
 *  Para ejecutar el programa, se debe realizar:
 *  make ejecutarlo 
 *  Si se quiere compilar sin el archivo make
 *  nvcc  -I/usr/local/cuda/include MCQR.cu -lcublas -lcusolver
 *  Rutina para calcular mínimos cuadrados usando la factorización QR
 */


#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include "definiciones.h"

double randomRange(double m,double n);

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;    
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    int m = 1;
    int lda = m;
    int ldb = m;
    int nrhs = 1; 
    arreglo_2d_T AA, BB, XX;

    // Lee matriz A


    m = atoi(argv[1]);

    AA = (arreglo_2d_T) malloc(sizeof(*AA));
    
    renglones(AA)=m;
    columnas(AA)=m;
    entradas(AA)=(double*)malloc(renglones(AA)*columnas(AA)*sizeof(double));
    inicializa_matriz(AA,"A.txt");
    imprime_matriz(AA);


    // Lee Matriz B

    printf("\n");

    BB = (arreglo_2d_T) malloc(sizeof(*BB));
    
    renglones(BB)=m;
    columnas(BB)=1;
    entradas(BB)=(double*)malloc(renglones(BB)*columnas(BB)*sizeof(double));
    inicializa_matriz(BB,"B.txt");
    imprime_matriz(BB);


    // Configura matriz con el resultado

    XX = (arreglo_2d_T) malloc(sizeof(*XX));

    renglones(XX)=m;
    columnas(XX)=1;
    entradas(XX)=(double*)malloc(renglones(XX)*columnas(XX)*sizeof(double));

    nrhs = 1;
    
    lda = m;
    ldb = m;    
    
    double A[lda*m]; 
    double B[ldb*nrhs];
    double XC[ldb*nrhs]; 
        

    double *d_A = NULL; 
    double *d_tau = NULL; 
    double *d_B  = NULL; 
    int    *devInfo = NULL; 
    double *d_work = NULL;
    int     lwork = 0; 
    int info_gpu = 0;
    const double one = 1;
    

    cusolver_status = hipsolverDnCreate(&cusolverH);    
    cublas_status = hipblasCreate(&cublasH);        
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(double) * m);
    cudaStat3 = hipMalloc ((void**)&d_B  , sizeof(double) * ldb * nrhs);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    cudaStat1 = hipMemcpy(d_A, entradas(AA), sizeof(double) * lda * m   , hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, entradas(BB), sizeof(double) * ldb * nrhs, hipMemcpyHostToDevice);
    cusolver_status = hipsolverDnDgeqrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork);
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    cusolver_status = hipsolverDnDgeqrf(cusolverH, m, m, d_A, lda, d_tau, d_work, lwork, devInfo);
    cudaStat1 = hipDeviceSynchronize();    
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    cusolver_status= hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,m, 
                                      nrhs, m, d_A, lda,d_tau,d_B,ldb,
                                      d_work,lwork,devInfo);
    cudaStat1 = hipDeviceSynchronize();    
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    cublas_status = hipblasDtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                                m,nrhs,&one,d_A,lda,d_B,ldb);
    cudaStat1 = hipDeviceSynchronize();    
    cudaStat1 = hipMemcpy(entradas(XX), d_B, sizeof(double)*ldb*nrhs, hipMemcpyDeviceToHost);


    printf("\n");        
    printf("=======================\n");        
    printf("Solución X\n");
    imprime_matriz(XX);
    printf("=======================\n");

    if (d_A    ) hipFree(d_A);
    if (d_tau  ) hipFree(d_tau);
    if (d_B    ) hipFree(d_B);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);


    if (cublasH ) hipblasDestroy(cublasH);   
    if (cusolverH) hipsolverDnDestroy(cusolverH);   

    hipDeviceReset();

    return 0;
}

double randomRange(double m,double n){
    return m + (rand() / ( RAND_MAX / (n-m) ) ) ;  
}
