#include "hip/hip_runtime.h"
/* Equipo 9
 *  Factorización QR utilizando CUDA, cuSolver
 *  Para compilar se debe ejecutar:
 *  make 
 *  Para ejecutar el programa, se debe realizar:
 *  make ejecutarlo 
 *  Si se quiere compilar sin el archivo make
 *  nvcc  -I/usr/local/cuda/include Qr.cu -lcublas -lcusolver
 *  Las matrices se alamacen en orden column major
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h> 
#include <hipsolver.h>


void imprimeMatriz(int m, int n, const double*A, int lda, const char* name);

int main(int argc, char*argv[])
{

    /* Se inicializan variables*/

    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;    
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;

    /* Se realiza una prueba con una matriz de 3 x 2  */

    int m = 3;     // Dimension de la matriz
    int n = 2; 
    int lda = m;   
    
    /* Se define la matriz , se va cambiar para que se lea la matriz de un archivo de texto*/ 

    double A[lda*n] = { 3.0, 4.0, 1.0, -2.0, 5.1, -1.2};
    double Q[lda*n]; 
    double R[n*n]; 
    
    double *d_A = NULL;
    double *d_tau = NULL;
    int    *devInfo = NULL;
    double *d_work = NULL;

    double *d_R = NULL;

    int info_gpu = 0;

    int lwork_geqrf = 0;
    int lwork_orgqr = 0;
    int lwork = 0;


    double h_one = 1;
    double h_minus_one = -1;
    
    printf("Matriz A:\n");
    imprimeMatriz(m, n, A, lda, "A");


    cusolver_status = hipsolverDnCreate(&cusolverH);
    cublas_status = hipblasCreate(&cublasH);

    cudaStat1 = hipMalloc ((void**)&d_A , sizeof(double)*lda*n);
    cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(double)*n);
    cudaStat3 = hipMalloc ((void**)&devInfo, sizeof(int));
    cudaStat4 = hipMalloc ((void**)&d_R , sizeof(double)*n*n);


    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*lda*n,hipMemcpyHostToDevice);
    
    cusolver_status = hipsolverDnDgeqrf_bufferSize(cusolverH,m,n,d_A,lda,&lwork_geqrf);                                                    
    //cusolver_status = hipsolverDnDorgqr_bufferSize(cusolverH,m,n,n,d_A,lda,&lwork_orgqr);
        
    lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);


    /* Calcula QR*/ 
    cusolver_status = hipsolverDnDgeqrf(cusolverH,m,n,d_A,lda,d_tau,d_work,lwork,devInfo);
    cudaStat1 = hipDeviceSynchronize();    

    /* Calcula Q */                                                       
    cusolver_status= hipsolverDnDorgqr(cusolverH,m,n,n,d_A,lda,d_tau,d_work,lwork,devInfo);
    cudaStat1 = hipDeviceSynchronize();

    /* Copia a host */
    cudaStat1 = hipMemcpy(Q, d_A, sizeof(double)*lda*n,hipMemcpyDeviceToHost);

    printf("Matriz Q:\n");
    imprimeMatriz(m, n, Q, lda, "Q");

    memset(R, 0, sizeof(double)*n*n);
    for(int j = 0 ; j < n ; j++){
        R[j + n*j] = 1.0; 
    }

    /* Copia de host al dispositivo */
    cudaStat1 = hipMemcpy(d_R, R, sizeof(double)*n*n, hipMemcpyHostToDevice);

    cublas_status = hipblasDgemm(cublasH,HIPBLAS_OP_T, HIPBLAS_OP_N, n, n,m, 
                                    &h_minus_one, d_A, lda,d_A, lda,&h_one,d_R,n);
    
    double dR_nrm2 = 0.0;
    cublas_status = hipblasDnrm2(cublasH, n*n, d_R, 1, &dR_nrm2);    


    // Liberar memoria 

    if (d_A)       hipFree(d_A);
    if (d_tau )    hipFree(d_tau);
    if (devInfo)   hipFree(devInfo);
    if (d_work )   hipFree(d_work);
    if (d_R)       hipFree(d_R);
    if (cublasH )  hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    hipDeviceReset();

    printf("Finaliza ejecución\n");

    return 0;
}



void imprimeMatriz(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
        printf("\n");
    }
}

