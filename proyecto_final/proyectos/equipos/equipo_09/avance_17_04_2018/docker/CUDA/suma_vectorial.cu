
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10
__global__ void suma_vect(int *a, int *b, int *c){
	int tid = blockIdx.x;
	if(tid<N)
		c[tid] = a[tid]+b[tid];
}
int main(void){
	int a[N], b[N],c[N];
	int *device_a, *device_b, *device_c;
	int i;
	//alojando en device
	hipMalloc((void **)&device_a, sizeof(int)*N);
	hipMalloc((void **)&device_b, sizeof(int)*N);
	hipMalloc((void **)&device_c, sizeof(int)*N);
	//llenando los arreglos:
	for(i=0;i<N;i++){
		a[i]=i;
		b[i]=i*i;
	}
	//copiamos arreglos a, b a la GPU
	hipMemcpy(device_a,a,N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_b,b,N*sizeof(int), hipMemcpyHostToDevice);
	//mandamos a llamar a suma_vect:
	suma_vect<<<N,1>>>(device_a,device_b,device_c);
	//copia del resultado al arreglo c:
	hipMemcpy(c,device_c,N*sizeof(int),hipMemcpyDeviceToHost);
	for(i=0;i<N;i++)
		printf("%d+%d = %d\n",a[i],b[i],c[i]);
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
	return 0;
}