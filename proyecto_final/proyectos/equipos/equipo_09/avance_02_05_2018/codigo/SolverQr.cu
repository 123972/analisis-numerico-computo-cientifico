#include "hip/hip_runtime.h"
/* Equipo 9
 *  Factorización QR utilizando CUDA, cuSolver
 *  Para compilar se debe ejecutar:
 *  make 
 *  Para ejecutar el programa, se debe realizar:
 *  make ejecutarlo 
 *  Si se quiere compilar sin el archivo make
 *  nvcc  -I/usr/local/cuda/include Solver.cu -lcublas -lcusolver
 *  Las matrices se alamacen en orden column major
 *  TODO: 
 *  Revisando la compilacion usando cuda + c
 *  https://devblogs.nvidia.com/separate-compilation-linking-cuda-device-code/
 */


#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include "definiciones.h"

void imprimeMatriz(int m, int n, const double*A, int lda, const char* name);
double randomRange(double m,double n);

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;    
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    int m = 3;
    int lda = m;
    int ldb = m;
    int nrhs = 2; 
    arreglo_2d_T AA, BB;


    srand(175840);


    m = atoi(argv[1]);
    nrhs = atoi(argv[2]);
    
    lda = m;
    ldb = m;    
    
    double A[lda*m]; 
    double B[ldb*nrhs];
    double XC[ldb*nrhs]; 
    
    for(int i =0;i< m * m ;i++)
        A[i] = randomRange(-5,5);

    for(int i =0;i<m*nrhs;i++)
        B[i] = randomRange(-5,5);

        

    double *d_A = NULL; 
    double *d_tau = NULL; 
    double *d_B  = NULL; 
    int    *devInfo = NULL; 
    double *d_work = NULL;
    int     lwork = 0; 
    int info_gpu = 0;
    const double one = 1;

    printf("Matriz A:\n");
    imprimeMatriz(m, m, A, lda, "A");
    printf("=======================\n");
    printf("Vector B:\n");
    imprimeMatriz(m, nrhs, B, ldb, "B");
    printf("=======================\n");

    cusolver_status = hipsolverDnCreate(&cusolverH);
    
    cublas_status = hipblasCreate(&cublasH);
    
    
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(double) * m);
    cudaStat3 = hipMalloc ((void**)&d_B  , sizeof(double) * ldb * nrhs);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    
    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m   , hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * ldb * nrhs, hipMemcpyHostToDevice);
    
    cusolver_status = hipsolverDnDgeqrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork);
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    cusolver_status = hipsolverDnDgeqrf(cusolverH, m, m, d_A, lda, d_tau, d_work, lwork, devInfo);
    cudaStat1 = hipDeviceSynchronize();    
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    
    cusolver_status= hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,m, 
                                      nrhs, m, d_A, lda,d_tau,d_B,ldb,
                                      d_work,lwork,devInfo);
    cudaStat1 = hipDeviceSynchronize();    

    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);


    cublas_status = hipblasDtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                                m,nrhs,&one,d_A,lda,d_B,ldb);
  cudaStat1 = hipDeviceSynchronize();

  cudaStat1 = hipMemcpy(XC, d_B, sizeof(double)*ldb*nrhs, hipMemcpyDeviceToHost);


  printf("Vector Solución X\n");
  imprimeMatriz(m, nrhs, XC, ldb, "X");
  printf("=======================\n");

  if (d_A    ) hipFree(d_A);
  if (d_tau  ) hipFree(d_tau);
  if (d_B    ) hipFree(d_B);
  if (devInfo) hipFree(devInfo);
  if (d_work ) hipFree(d_work);


  if (cublasH ) hipblasDestroy(cublasH);   
  if (cusolverH) hipsolverDnDestroy(cusolverH);   

  hipDeviceReset();

  return 0;
}


void imprimeMatriz(int m, int n, const double*A, int lda, const char* name){
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s[%d,%d] = %f\t", name, row+1, col+1, Areg);
        }
        printf("\n");
    }
}


double randomRange(double m,double n){
    return m + (rand() / ( RAND_MAX / (n-m) ) ) ;  
}




