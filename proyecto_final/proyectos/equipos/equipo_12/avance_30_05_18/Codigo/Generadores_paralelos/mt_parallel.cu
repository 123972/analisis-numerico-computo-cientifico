
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>


#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void generate_kernel(hiprandStateMtgp32 *state, 
                                int n,
                                int *result)
{
    int id = threadIdx.x + blockIdx.x * 256;
    int count = 0;
    unsigned int x;
    for(int i = 0; i < n; i++) {
        x = hiprand(&state[blockIdx.x]);
        if(x & 1) {
            count++;
        }
    }

    result[id] += count;
}

int main(int argc, char *argv[])
{
    int i;
    long long total;
    hiprandStateMtgp32 *devMTGPStates;
    mtgp32_kernel_params *devKernelParams;
    int *devResults, *hostResults;
    int sampleCount = 10000;
    
   
    if (argc == 2) {
        sscanf(argv[1],"%d",&sampleCount);
    }
        

    hostResults = (int *)calloc(64 * 256, sizeof(int));

  
    CUDA_CALL(hipMalloc((void **)&devResults, 64 * 256 * 
              sizeof(int)));

  
    CUDA_CALL(hipMemset(devResults, 0, 64 * 256 * 
              sizeof(int)));

  
    CUDA_CALL(hipMalloc((void **)&devMTGPStates, 64 * 
              sizeof(hiprandStateMtgp32)));
    
 
    CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
    
   
    /* copia de parametros a memoria del device              */
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
    
    /* Inicualiza thread blocls */
    CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, 
                mtgp32dc_params_fast_11213, devKernelParams, 64, 1234)); 
 
    
    for(i = 0; i < 10; i++) {
        generate_kernel<<<64, 256>>>(devMTGPStates, sampleCount, devResults);
    }

    /* copia al host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 256 * 
        sizeof(int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 1000; i++) {

        total += hostResults[i];
        printf("%u\n", hostResults[i]);
    }

   
    

    /* Cleanup */
    CUDA_CALL(hipFree(devMTGPStates));
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    printf("^^^^ kernel_mtgp_example PASSED\n");
    return EXIT_SUCCESS;
}