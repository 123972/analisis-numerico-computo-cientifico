#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 1000

#define MAX 100

/* Inicializo valores iniciales*/
__global__ void init(unsigned int seed, hiprandState_t* states) {

  
  hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

/* Pone un número aleatorio en un vector de números */
__global__ void randoms(hiprandState_t* states, unsigned int* numbers) {
  
  numbers[blockIdx.x] = hiprand(&states[blockIdx.x]) % 100000; /* este último número es el largo del número */
}

int main( ) {
  /* mandamos un state a cada thread  */
  hiprandState_t* states;

  hipMalloc((void**) &states, N * sizeof(hiprandState_t));

  /* inicializamos todos los threads */
  init<<<N, 1>>>(time(0), states);

  unsigned int cpu_nums[N];
  unsigned int* gpu_nums;
  hipMalloc((void**) &gpu_nums, N * sizeof(unsigned int));

  /* llamamos la función para generar números */
  randoms<<<N, 1>>>(states, gpu_nums);

  hipMemcpy(cpu_nums, gpu_nums, N * sizeof(unsigned int), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    printf("%u\n", cpu_nums[i]);
  }

  hipFree(states);
  hipFree(gpu_nums);

  return 0;
}