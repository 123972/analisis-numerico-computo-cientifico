#include "hip/hip_runtime.h"
/************************************************************************************************/
/************ Arturo Gonzalez Bencomo 172906, Arturo Torre Gonzalez 90226 ***********************/
/************************************************************************************************/
/********************************** MatrixMul.cu ************************************************/
/************************************************************************************************/
/** Este programa escrito en c++ hace una multiplicacion matricial de forma paralela utilizando */
/** cuda. ***************************************************************************************/


/*Se importan librerias*/

#include <iostream>            //Equivalente a stdio.h
#include <vector>              //Clase vector que permite crear arrays dinamicos.
#include <stdlib.h>            //stdlib de c
#include <time.h>              //Manejo de funciones de tiempo
#include <hip/hip_runtime.h>      //Administrador del runtime de cuda
#include "kernel.h"            //Importamos el archivo de prototipo del kernel
#include "kernel.cu"           //Implementacion del kernel
#include "dev_array.h"         //Estructura de datos tipo array
#include <math.h>              //Funciones matematicas, la necesitamos para rellenar la matriz inicial 

using namespace std;

int main()
{
    // Este Programa llevara a cabo la multiplicacion matricial A*B y asignara el resultado a C    
    int N = 16;                                 
    int SIZE = N*N;                               //Tamanio de las matrices 16x16 

    // Asignamos el espacio en memoria de las matrices en la memoria principal
    vector<float> h_A(SIZE);
    vector<float> h_B(SIZE);
    vector<float> h_C(SIZE);

    // Inicializamos las matrices dentro del programa principal con valores iniciales de seno y coseno (funciones de la libreria math.h) 
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = sin(i);
            h_B[i*N+j] = cos(i);
        }
    }

    // Reservamos el espacio en memoria en el GPU
    dev_array<float> d_A(SIZE);
    dev_array<float> d_B(SIZE);
    dev_array<float> d_C(SIZE);

    // Copiamos las variables en memoria principal a memoria del device
    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);

    //Mandamos llamar la funcion matrixMultiplication definida en el archivo kernel.cu, que es la que lleva a cabo el procesamiento paralelo.
    matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
    
    //
    hipDeviceSynchronize();

    d_C.get(&h_C[0], SIZE);
    hipDeviceSynchronize();


    // Imprimimos matrices originales y salida para comprobar la operacion
    // h_A
    cout << "MATRIZ A\n";    
    for (int ROW=0; ROW < N; ROW++){
        for (int COL=0; COL < N; COL++){
            cout << h_A[ROW * N + COL];
            cout << " ";
        }
        cout << "\n";
    }

    // h_B    
    cout << "MATRIZ B\n";
    for (int ROW=0; ROW < N; ROW++){
        for (int COL=0; COL < N; COL++){
            cout << h_B[ROW * N + COL];
            cout << " ";
        }
        cout << "\n";
    }

    // h_C
    cout << "MATRIZ C\n";
    for (int ROW=0; ROW < N; ROW++){
        for (int COL=0; COL < N; COL++){
            cout << h_C[ROW * N + COL];
            cout << " ";
        }
        cout << "\n";
    } 
    
    return 0;
}
