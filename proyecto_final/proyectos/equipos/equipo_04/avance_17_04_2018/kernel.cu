#include "hip/hip_runtime.h"
/************************************************************************************************/
/************ Arturo Gonzalez Bencomo 172906, Arturo Torre Gonzalez 90226 ***********************/
/************************************************************************************************/
/********************************** kernel.cu ************************************************/
/************************************************************************************************/
/** Este programa especifica como tal la funcion matrixMultiplication, y el kernel que es como **/
/** tal el codigo ejecutado en el GPU ***********************************************************/


/*Importamos librerias*/
#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdlib.h>

using namespace std;


/*Esta funcion es el kernel que ejecuta la multiplicacion en paralelo*/
__global__ 
void matrixMultiplicationKernel(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}


/* Esta funcion prepara los parametros del kernel, se especifica 1 grid de 1 bloque asi como */
/* 256 threads por bloque que corresponden al total de operaciones producto punto. ***********/
void matrixMultiplication(float *A, float *B, float *C, int N){

    // Estructuras de datos tipo dim3    
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
        if (N*N > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
        }

    //Este renglon inicializa el computo en paralelo en la GPU.
    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, N);
}

