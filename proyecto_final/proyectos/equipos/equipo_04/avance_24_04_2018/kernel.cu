#include "hip/hip_runtime.h"
/************************************************************************************************/
/************ Arturo Gonzalez Bencomo 172906, Arturo Torre Gonzalez 90226 ***********************/
/************************************************************************************************/
/********************************** kernel.cu ***************************************************/
/************************************************************************************************/
/** Este programa especifica como tal la funcion para convertir de rgb a grayscale **************/
/** tal el codigo ejecutado en el GPU ***********************************************************/


/*Importamos librerias*/
#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdlib.h>

using namespace std;

/*Esta funcion es el kernel que ejecuta grayscale en paralelo sobre cada pixel de la matriz*/
__global__
void grayscale(float *RED, float *GREEN, float *BLUE, float *GRAY){
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            // Esta funcion es la que lleva a cabo la transformacion de colores rgb a blanco y negro
            GRAY[y][x]  = truncf(0.2989*px[0] + 0.587*px[1] + 0.114*px[2]);
        }
    }
    C[ROW * N + COL] = tmpSum;
}

