#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <vector>

//includes de CUDA 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "hip/device_functions.h"

//Clases necesarias
#include "BinaryRestriction.h"
#include "VectorBR.h"
#include "BinaryMixer.h"

//__device__ int cont = 0;
int block_width = 1024;



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: \"%s\": %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//funcion que me regrese el contador

__global__ void ParallelMix(unsigned long long int * d_cont ,unsigned int number_threads, int d_rest_size ,int numStations, unsigned int * d_temp, unsigned int * d_rest, unsigned int * d_out) {

	//ID del thread
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	//Condition that verifies the usefulness of the thread.
	if (tid < number_threads) {

		//VARIABLES

		//Esta variable auxiliar denota la doble asignación de una actividad si es >0.  
		unsigned int aux_v = 0;

		//Esta otra variable denota que actividades se hana asignado.
		unsigned int cum_v = 0;

		//Bandera que se dispara si una actividade se ha asigando dos veces.
		bool flag = false;

		//Este vector auxiliar es una cota, segun el enumero de estaciones de un problema.
		//Por el momento dejarlo en 7. (revisar despues con el doc)
		unsigned int aux[7] = { 0,0,0,0,0,0,0 };

		//Imprimimos el thread
		//printf("Thread Num: %d \n", number_threads);
		//printf("Thread ID: %d \n", tid);

		//for donde se itern sobre todas las tablas de solución en Ri.
		//Cada thread intersectará cada tabla de solución en la nueva restriccion d_temp.
		for (int i = 0; i <d_rest_size ; i++) {

			//Perform OR operations on every line (STATION) of the solution table.
			//Se hace un or en cada linea (osea cad estación) de la tabla de solución.

			//Esto para identificar si una actividad esta asignada a más de una estación.
			for (int k = 0; k<numStations; k++) {

				// El pseudo código del algoritmo de intersección booleana es el siguiente.
				//1. Perform OR operation on table row (k) from tid solution table (d_temp) and new constraint (d_rest).
				//1. Primero haz un or sobre la fila k (osea estación) de la tabla de solución d_temp y la nueva restriccioń d_rest.
				//	 Se guarda el resultado en aux[k].cx, que declaramos arriba.
				//		1.1. Realizar operación AND con aux[k] y cum_v. 
				///////////////////////////////////////////////////////////////////////////////

				aux[k] = *(d_temp + (tid*numStations + k)) | *(d_rest + k + (i*numStations));


				//actualización de vector aux con el AND siempre y cuando la bandera no se
				//haya disparado.
				if (!flag) {
					aux_v = cum_v & aux[k];
					if (aux_v > 0) {
						flag = true;
					}
					cum_v = cum_v^aux[k];
				}
			}

			if (!flag) {
				//Tabla VALIDA
				aux_v = atomicAdd(d_cont, 1);
				
				for (int j = 0; j < numStations; j++) {
					*(d_out + (aux_v*numStations) + j) = aux[j];
					
				}
				printf("VALIDA %d, %d\n", tid, i);
				flag = false;
				aux_v = 0;
				cum_v = 0;

			}
			else {
				//Tabla INVALIDA
				printf("INVALIDA %d, %d\n", tid, i);
				flag = false;
				aux_v = 0;
				cum_v = 0;

			}
		}
		//Termina forloop
		//free memory from the variables

	}
}

extern "C" bool
runTest(int argc, const char **argv, unsigned long long int * h_cont, int numStations, unsigned int * h_temp, unsigned long long int h_temp_size , unsigned int * h_rest, int h_rest_size, unsigned int * h_out) {

	//use command-line specified CUDA device, otherwise use device with highest Gflops
	//esta linea tal vez no sea necesaria, al fin sólo tomará la GPU del servidor.
	findCudaDevice(argc, (const char **)argv);

	//Calcula los threads y los bloques.
	unsigned int num_threads = h_temp_size / numStations;
	unsigned int num_blocks = 1 + (num_threads / block_width);
	
	cout << "Number of threads: " << num_threads << endl;
	cout << "Number of Blocks:  " << num_blocks << endl;

	//GPU memory status variables
	size_t total_mem, free_mem;

	//Memory requirement for restriction
	const size_t mem_size_rest = sizeof(int) * h_rest_size;

	//Create Device Variables
	unsigned int * d_rest;
	unsigned int * d_out;
	unsigned long long int * d_cont;
	unsigned long long int * internal_cont = new unsigned long long int[1];
	*(internal_cont) = 0;


	//Se hacen las reservas de memoria en la gpu.
	checkCudaErrors(hipMalloc((void **)&d_rest, mem_size_rest));
	checkCudaErrors(hipMalloc((void **)&d_cont, sizeof(unsigned long long int)));

	//se copia de lamemoria el host al device.
	checkCudaErrors(hipMemcpy(d_rest, h_rest, mem_size_rest, hipMemcpyHostToDevice));
	
	gpuErrchk(hipMemcpy(d_cont, h_cont, sizeof(unsigned long long int), hipMemcpyHostToDevice));

	//reporte de la memoria.
	hipMemGetInfo(&free_mem, &total_mem);
	printf("Free Memory: %s \tTotal Memory: %s \n", to_string(free_mem), to_string(total_mem));

	
	//cout << "Out vector byte size: " << to_string(mem_size_out) << endl;
	//cout << "Rest vector size: " << h_rest_size << endl;
	//printf("Out vector size: %s\n", to_string((h_temp_size * (h_rest_size / numStations))));
	

	//Declare a 1 MB constant
	const size_t MB = 1 << 20;

	//Buffer Conffiguration: Servirá para administrar los batches en la GPU.
	hipError_t status;
	unsigned int * buffer;

	//Allocate at mmost 500MB of memory for the buffer size.
	//This limitation will allow us to allocate enough memory to store the resulting solutions vector.
	size_t buffer_size = std::min((long long)4294967296 / (h_rest_size / numStations), (long long)536870912);
        //Make sure the allocation is succesful
	for (; buffer_size > MB; buffer_size -= MB) {
		//Allocate memory for buffer
		status = hipMalloc((void **)&buffer, buffer_size);
		if (status == hipSuccess)
			break;
	}

	//Bytes reservados en la GPU
	cout << "Allocated " << buffer_size << " bytes on GPU. " << endl;
	cout << h_temp_size << " int items require processing. " << endl;
	//PRINT: Batch size.

	
	//Tamaño del buffer auxiliar, ayua a preservar la integridad del vector de soluciones.
	size_t aux_buff_size = (buffer_size / sizeof(int)) % numStations;

	//Numero de elementos enteros que serán procesados en este batch.
	size_t batchN = (buffer_size - aux_buff_size) / sizeof(int);

	//elementos a la espera de procesarse.
	size_t remainN = h_temp_size;

	
	int elements_out = batchN * (h_rest_size / numStations);
	const size_t mem_size_out = elements_out * sizeof(int);
	checkCudaErrors(hipMalloc((void **)&d_out, mem_size_out));

	cout << "Number of int proccessed in this batch: " << batchN << endl;


	//Loop sobre los datos del host, en batches prdefinidos arriba.
	for (; remainN > 0; remainN -= batchN) {

		cout << "\n BATCH \n\n";

		//Si remain es menor que el tamano del batch,
		//ajusto el tamano del batch a lo que queda de datos
		//y en caso contrario lo dejo igual
		batchN = (remainN<batchN) ? remainN : batchN;
		size_t worksize = batchN * sizeof(int);
		cout << "Processing " << batchN << "/" << remainN << " items remaining." << endl;

		//COPY values from host temporal solution to buffer.
		//Process the items in buffer.
		hipMemcpy(buffer, h_temp, worksize, hipMemcpyHostToDevice);

		//La cantidad de threads se determina por el número de tablas de solucion en el vector.
		num_threads = batchN / numStations;
		num_blocks = 1 + (num_threads / block_width);
		//Prints
		cout << "Number of threads: " << num_threads << endl;
		cout << "Number of Blocks:  " << num_blocks << endl;

		

		//Lanzamos el kernel.
		ParallelMix << < num_blocks, block_width >> >(d_cont, num_threads, h_rest_size / numStations, numStations, buffer, d_rest, d_out);


		cout << "Succesful execution\n";

		//copiamos el contador de tablas de lgpu al cpu.
		gpuErrchk(hipMemcpy(h_cont, d_cont, sizeof(unsigned long long int), hipMemcpyDeviceToHost));

		*(internal_cont) += *(h_cont);
		cout << "Succesful execution\n";
		//Se copin las soluciones validas al host.
		//d_cont es el numero de soluciones de tablas válidas  a partir ed que se corrió el kernel.
		
		cout << *(h_cont) << endl;
		cout << "ACUM: " << *(internal_cont) << endl;

		gpuErrchk(hipMemcpy(h_out, d_out,*(h_cont)*numStations*sizeof(int), hipMemcpyDeviceToHost));
		
		//Se actualizan apuntadores en la magnitud del batch que se procesó.
		h_temp += batchN;
		//Se actualiza el apuntado h_put en la magnitud del numero de soluciones validas del batch procesado.
		h_out = h_out + (*(h_cont)*numStations);

		gpuErrchk(hipMemset(d_cont, (unsigned long long int)0, sizeof(unsigned long long int)));
		gpuErrchk(hipMemset(d_out, 0,mem_size_out));
	}

	

	//GPU memory status REPORT.
	hipMemGetInfo(&free_mem, &total_mem);
	printf("Free Memory: %s \tTotal Memory: %s \n", to_string(free_mem), to_string(total_mem));

	*(h_cont) = *(internal_cont);


	//cleanup device memory

	//Reset Device
	hipDeviceReset();

	return 0;


}
