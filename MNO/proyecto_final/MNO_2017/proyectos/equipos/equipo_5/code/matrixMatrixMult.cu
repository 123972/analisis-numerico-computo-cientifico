//#include<stdio.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void gaxpymm(int *y, int *a, int *b, int m, int n, int p){
	int bid = blockIdx.x;
        int tid = threadIdx.x;
	extern __shared__ int dots_s[];
	if(bid<m)
		if(tid<n){
			for(int c=0;c<p;c++)
				dots_s[bid*n*p+tid*p+c] = a[bid*n+tid] * *(b+(tid*p+c));
	__syncthreads();
	if(tid == 0){
		for(int c=0;c<p;c++)
			for(int i=1;i<n;i++){
				dots_s[bid*n*p+c] +=dots_s[bid*n*p+i*p+c];
//			printf("y=%d, dots_s=%d, bid=%d, tid=%d, i=%d, n=%d\n",dots_s[bid*n], dots_s[bid*n+i],bid,tid,i,n);
			}
		for(int c=0;c<p;c++)
			*(y+(bid*p+c))=dots_s[bid*n*p+c];
//		printf("y[%d]=%d, bid=%d, tid=%d\n",bid,y[bid],bid,tid);
		}
	}
}

std::vector<int> matrixMatrixMultiplication(int* a, int* b, int mc, int nc, int pc){
	int* m, *n, *p;
	m = &mc;
	n = &nc;
	p = &pc;
	std::vector<int> y(mc*pc,0);
	int *device_y, *device_a, *device_b;
	int *device_m, *device_n, *device_p;
	//alojando en device
	hipMalloc((void **)&device_y, sizeof(int)*mc*pc);
	hipMalloc((void **)&device_a, sizeof(int)*mc*nc);
	hipMalloc((void **)&device_b, sizeof(int)*nc*pc);
	hipMalloc((void **)&device_m, sizeof(int));
	hipMalloc((void **)&device_n, sizeof(int));
	hipMalloc((void **)&device_p, sizeof(int));
	//copiamos arreglos a, x a la GPU
	hipMemcpy(device_a,a,mc*nc*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_b,b,nc*pc*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_y,y.data(),mc*pc*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_m,m,sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_n,n,sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_p,p,sizeof(int), hipMemcpyHostToDevice);
	//mandamos a llamar a suma_vect:
	gaxpymm<<<mc,nc,sizeof(int)*mc*nc*pc>>>(device_y,device_a,device_b,mc,nc,pc);
//	for(unsigned i=0; i<y.size();i++)
//		std::cout << "yi[i] = " << y[i] << "\n";
	//copia del resultado al arreglo y:
	hipMemcpy(y.data(),device_y,mc*pc*sizeof(int),hipMemcpyDeviceToHost);
//	for(unsigned i=0; i<y.size();i++)
//		std::cout << "yf[i] = " << y[i] << "\n";
	hipFree(device_y);
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_m);
	hipFree(device_n);
	hipFree(device_p);
	return y;
}

