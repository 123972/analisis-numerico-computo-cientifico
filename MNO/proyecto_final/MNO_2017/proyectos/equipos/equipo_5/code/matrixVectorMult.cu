//#include<stdio.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void gaxpy(int *y, int *a, int *x, int m, int n){
	int bid = blockIdx.x;
        int tid = threadIdx.x;
	extern __shared__ int dots_s[];
	if(bid<m)
		if(tid<n){

			dots_s[bid*n+tid] = a[bid*n+tid] * *(x+tid);
	__syncthreads();
	if(tid == 0){
		for(int i=1;i<n;i++){
			dots_s[bid*n] +=dots_s[bid*n+i];
//			printf("y=%d, dots_s=%d, bid=%d, tid=%d, i=%d, n=%d\n",dots_s[bid*n], dots_s[bid*n+i],bid,tid,i,n);
		}
		*(y+bid)=dots_s[bid*n];
//		printf("y[%d]=%d, bid=%d, tid=%d\n",bid,y[bid],bid,tid);
		}
	}
}

std::vector<int> matrixVectorMultiplication(int* a, int mc, int nc, int* x){
	int* m, *n;
	m = &mc;
	n = &nc;
	std::vector<int> y(mc,0);
	int *device_y, *device_a, *device_x;
	int *device_m, *device_n;
	//alojando en device
	hipMalloc((void **)&device_y, sizeof(int)*mc);
	hipMalloc((void **)&device_a, sizeof(int)*mc*nc);
	hipMalloc((void **)&device_x, sizeof(int)*nc);
	hipMalloc((void **)&device_m, sizeof(int));
	hipMalloc((void **)&device_n, sizeof(int));
	//copiamos arreglos a, x a la GPU
	hipMemcpy(device_a,a,mc*nc*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_x,x,nc*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_y,y.data(),mc*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_m,m,sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_n,n,sizeof(int), hipMemcpyHostToDevice);
	//mandamos a llamar a suma_vect:
	gaxpy<<<mc,nc,sizeof(int)*mc*nc>>>(device_y,device_a,device_x,mc,nc);
//	for(unsigned i=0; i<y.size();i++)
//		std::cout << "yi[i] = " << y[i] << "\n";
	//copia del resultado al arreglo y:
	hipMemcpy(y.data(),device_y,mc*sizeof(int),hipMemcpyDeviceToHost);
//	for(unsigned i=0; i<y.size();i++)
//		std::cout << "yf[i] = " << y[i] << "\n";
	hipFree(device_y);
	hipFree(device_a);
	hipFree(device_x);
	hipFree(device_m);
	hipFree(device_n);
	return y;
}

