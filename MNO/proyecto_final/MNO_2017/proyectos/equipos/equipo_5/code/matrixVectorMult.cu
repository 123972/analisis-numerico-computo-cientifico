//#include<stdio.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void gaxpy(double *y, double *a, double *x, int m, int n){
	int bid = blockIdx.x;
        int tid = threadIdx.x;
	extern __shared__ double dots_s[];
	if(bid<m)
		if(tid<n){

			dots_s[bid*n+tid] = a[bid*n+tid] * *(x+tid);
	__syncthreads();
	if(tid == 0){
		for(int i=1;i<n;i++){
			dots_s[bid*n] +=dots_s[bid*n+i];
//			printf("y=%d, dots_s=%d, bid=%d, tid=%d, i=%d, n=%d\n",dots_s[bid*n], dots_s[bid*n+i],bid,tid,i,n);
		}
		*(y+bid)=dots_s[bid*n];
//		printf("y[%d]=%d, bid=%d, tid=%d\n",bid,y[bid],bid,tid);
		}
	}
}

std::vector<double> matrixVectorMultiplication(double* a, int mc, int nc, double* x){
	int* m, *n;
	m = &mc;
	n = &nc;
	std::vector<double> y(mc,0);
	double *device_y, *device_a, *device_x;
	int *device_m, *device_n;
	//alojando en device
	hipMalloc((void **)&device_y, sizeof(double)*mc);
	hipMalloc((void **)&device_a, sizeof(double)*mc*nc);
	hipMalloc((void **)&device_x, sizeof(double)*nc);
	hipMalloc((void **)&device_m, sizeof(int));
	hipMalloc((void **)&device_n, sizeof(int));
	//copiamos arreglos a, x a la GPU
	hipMemcpy(device_a,a,mc*nc*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_x,x,nc*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_y,y.data(),mc*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_m,m,sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_n,n,sizeof(int), hipMemcpyHostToDevice);
	//mandamos a llamar a suma_vect:
	gaxpy<<<mc,nc,sizeof(double)*mc*nc>>>(device_y,device_a,device_x,mc,nc);
//	for(unsigned i=0; i<y.size();i++)
//		std::cout << "yi[i] = " << y[i] << "\n";
	//copia del resultado al arreglo y:
	hipMemcpy(y.data(),device_y,mc*sizeof(double),hipMemcpyDeviceToHost);
//	for(unsigned i=0; i<y.size();i++)
//		std::cout << "yf[i] = " << y[i] << "\n";
	hipFree(device_y);
	hipFree(device_a);
	hipFree(device_x);
	hipFree(device_m);
	hipFree(device_n);
	return y;
}

